#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void Transpose(unsigned *matrix,unsigned *result ,unsigned size){
    
    unsigned id =blockDim.x*blockIdx.x+threadIdx.x;
    unsigned i=id/size;
    unsigned j=id%size;
    for(unsigned k=0;k<size;k++){
    result[i*size+j]=matrix[j*size+i];
 }
} // end Transpose

void initmatrix(unsigned *matrix,unsigned size){
        for(unsigned i=0;i<size;i++){
            for(unsigned j=0;j<size;j++){
                                        matrix[i*size+j]=(i*size+j);
                                        }
                                    }
}// end of initmatrix

void reset(unsigned *matrix,unsigned size){
     for(unsigned i=0;i<size;i++){
        for(unsigned j=0;j<size;j++){
                                    matrix[i*size+j]=0;
                                    }
                                }
}// end of reset

void printMatrix(unsigned *matrix,unsigned size){
    for(unsigned i=0;i<size;i++){
        for(unsigned j=0;j<size;j++){
                                    printf("%8d",matrix[i*size+j]);
                                    }
        printf("\n");
    }
}

#define N 64
#define BLCOKSIZE N
int main(){
 // host var
 unsigned *hmatrix,*hres;

 //device var
  unsigned *matrix,*res;

//allocate memory --host
hmatrix=(unsigned*)malloc(N*N*sizeof(unsigned));
hres=(unsigned*)malloc(N*N*sizeof(unsigned));

//allocate memory --device
hipMalloc(&matrix,N*N*sizeof(unsigned));
hipMalloc(&res,N*N*sizeof(unsigned));

//init 
initmatrix(hmatrix,N);
reset(hres,N);

//printing input matrix
printf("\nHost input\n");
printMatrix(hmatrix,N);


//cudamemcpy trasnfer host--> device
hipMemcpy(matrix,hmatrix,N*N*sizeof(unsigned),hipMemcpyHostToDevice);
hipMemcpy(res,hres,N*N*sizeof(unsigned),hipMemcpyHostToDevice);

Transpose<<<N,N>>>(matrix,res,N);
hipDeviceSynchronize();

//cudamem trasnfer device -->host
hipMemcpy(hres,res,N*N*sizeof(unsigned),hipMemcpyDeviceToHost);

printf("\nDevice output\n");
printMatrix(hres,N);

return 0;
}
