#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void Mykernel(int * count) {
printf("\n counter from device = %d", ++*count);
}

int main() {
int *count = 0;

// use pinned memory
hipHostAlloc(&count,sizeof(int),0);

//launch kernel
Mykernel<<<1,1>>>(count);
hipDeviceSynchronize();

//host can access count variable
printf("\ncounter from Host = %d", ++*count);
return 0;
}












































































