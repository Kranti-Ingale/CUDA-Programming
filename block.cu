#include<stdio.h>
#include<hip/hip_runtime.h>
#define BLOCKSIZE 1024

__global__ void mykernel(){

__shared__ unsigned s; // one s copy per block
int id = threadIdx.x; // blockDim.x*blockIdx.x+threadIdx.x

if(blockIdx.x==0){
    if(id==0) s=0;
    if(id==2) printf("\n block id= %d s = %d id = %d",blockIdx.x,s,id);
}
else{ //blockidx==1
    if(id==5) s=5;
    if(id==19) printf("\n block id= %d s = %d id = %d",blockIdx.x,s,id);
}

}
int main(){
    mykernel<<<2,BLOCKSIZE>>>(); // 2*1024
    hipDeviceSynchronize();
    return 0;
}