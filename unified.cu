#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(unsigned *a, unsigned *b, unsigned *c, unsigned N) {
    // Calculate global thread thread ID
    int id = blockDim.x * blockIdx.x + threadIdx.x;

    // Boundary check
    if (id < N) {
        c[id] = a[id] + b[id];
    }
} // end of vectorAdd

void init(unsigned *Arr,unsigned size){
    for(unsigned i=0;i<size;i++){
        Arr[i] = rand() % 100;
    }
} // end of init

void print(unsigned * a,unsigned *b,unsigned *c, unsigned N)
{
    printf("\n\tid\ta[id]\t+\tb[id]\t=\tc[id]");
    printf("\n--------------------------------------------------------------------");
    for (int i = 0; i < N; i++) {
         printf("\n\t%d\t%d\t+\t%d\t=\t%d",i,a[i],b[i],c[i]);
    }
} // end of print

#define N 32
#define BLOCK_SIZE 1024

int main(){
    // Declare unified memory pointers
    unsigned *a, *b, *c;

    // Allocation memory for these pointers
    hipMallocManaged(&a, N*sizeof(unsigned));
    hipMallocManaged(&b, N*sizeof(unsigned));
    hipMallocManaged(&c, N*sizeof(unsigned));

    // Initialize vectors 
    init(a,N);
    init(b,N);
 
    // CTAs per Grid
    int GRID_SIZE = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
   
    // Call CUDA kernel
    vectorAdd<<<GRID_SIZE, BLOCK_SIZE>>>(a, b, c, N);
    hipDeviceSynchronize();

    // print the result on the CPU
    print(a,b,c,N);
 
    // Free unified memory (same as memory allocated with cudaMalloc)
    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;
   
}




































































































