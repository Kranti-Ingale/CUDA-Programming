#include<stdio.h>
#include<hip/hip_runtime.h>

#define N 32
#define BLOCKSIZE 1024

__global__ void Even() {
    int id=blockDim.x*blockIdx.x+threadIdx.x;
    if(id<N){
                if(id%2 == 0)
                printf("\n%d",id);
            }
} // end of Even

__global__ void Odd() {
    int id=blockDim.x*blockIdx.x+threadIdx.x;
    if(id<N){
                 if(id%2 == 1)
                 printf("\n%d",id);
            }
} //end of odd


int main(){

    printf("\nEven");
    Even<<<(N+BLOCKSIZE-1)/BLOCKSIZE,BLOCKSIZE>>>();
    hipDeviceSynchronize();

    printf("\nOdd");
    Odd<<<(N+BLOCKSIZE-1)/BLOCKSIZE,BLOCKSIZE>>>();
    hipDeviceSynchronize();

    return 0;
}