#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void arr(int *a){
    a[threadIdx.x]=0;
}

#define N 32 //  <1024
int main(){
    int a[N],*da; //variable declaration
 
    // allocate device memory
    hipMalloc(&da,N*sizeof(int));

    //copy data host --> device
    hipMemcpy(da,a,N*sizeof(int),hipMemcpyHostToDevice);
  
    //launch kernel
    arr<<<1,N>>>(da);

    //copy data device --> Host
    hipMemcpy(a,da,N*sizeof(int),hipMemcpyDeviceToHost);

    //print output
    for(int i=0;i<N;i++){
        printf("\n a[%d] = %d",i,a[i]);
    }
    hipFree(da);
    return 0;
}